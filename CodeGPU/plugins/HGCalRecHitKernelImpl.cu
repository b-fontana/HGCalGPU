#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "HGCalRecHitKernelImpl.cuh"

__global__
void ee_step1(HGCUncalibratedRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGCeeUncalibratedRecHitConstantData cdata, size_t length)
{
  //dynamic shared memory
  extern __shared__ double s[];
  double   *sd = s;
  float    *sf = (float*)(sd + cdata.ndelem);
  uint32_t *su = (uint32_t*)(sf + cdata.nfelem);
  bool     *sb = (bool*)(su + cdata.nuelem);
  
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  //setting shared memory
  sd[0] = cdata.hgcEE_keV2DIGI_;
  sd[1] = cdata.hgceeUncalib2GeV_;
  for(unsigned int i=0; i < cdata.s_hgcEE_fCPerMIP_; ++i)
    sd[i+2] = cdata.hgcEE_fCPerMIP_[i];
  for(unsigned int i=0; i < cdata.s_hgcEE_cce_; ++i)
    sd[i+2+cdata.s_hgcEE_fCPerMIP_] = cdata.hgcEE_cce_[i];
  for(unsigned int i=0; i < cdata.s_hgcEE_noise_fC_; ++i)
    sd[i+2+cdata.s_hgcEE_fCPerMIP_+cdata.s_hgcEE_cce_] = cdata.hgcEE_noise_fC_[i];
  for(unsigned int i=0; i < cdata.s_rcorr_; ++i)
    sd[i+2+cdata.s_hgcEE_fCPerMIP_+cdata.s_hgcEE_cce_+cdata.s_hgcEE_noise_fC_] = cdata.rcorr_[i];

  for(unsigned int i=0; i < cdata.s_weights_; ++i)
    sf[i] = cdata.weights_[i];

  su[0] = cdata.rangeMatch_;
  su[1] = cdata.rangeMask_;

  sb[0] = cdata.hgcEE_isSiFE_;

  __syncthreads();

  if (tid==0)
      printf("%f %d %d\n", sd[2], su[1], sb[0]);
  for (unsigned int i = tid; i < length; i += blockDim.x * gridDim.x)
    {
      dst_soa.amplitude[i] = src_soa.amplitude[i];
    }
}

__global__
void hef_step1(HGCUncalibratedRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGChefUncalibratedRecHitConstantData cdata, size_t length)
{
}

__global__
void heb_step1(HGCUncalibratedRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGChebUncalibratedRecHitConstantData cdata, size_t length)
{
}

__global__
void ee_to_rechit(HGCRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGCeeUncalibratedRecHitConstantData cdata, size_t length)
{
  for (size_t i = blockDim.x * blockIdx.x + threadIdx.x; i < length; i += blockDim.x * gridDim.x)
    {
      dst_soa.energy[i] = 2.;
    }
}

__global__
void hef_to_rechit(HGCRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGChefUncalibratedRecHitConstantData cdata, size_t length)
{
}

__global__
void heb_to_rechit(HGCRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGChebUncalibratedRecHitConstantData cdata, size_t length)
{
}

/*
=======
>>>>>>> b5bfc7e2f47f926abb3dcd21cdf5e2094e53dd3f
//declared as extern in DataFormats/CaloRecHit/interface/CaloRecHit.h
#ifdef __CUDA_ARCH__
__constant__ uint32_t calo_rechit_masks[] = {0x00000000u, 0x00000001u, 0x00000003u, 0x00000007u, 0x0000000fu, 0x0000001fu,
					     0x0000003fu, 0x0000007fu, 0x000000ffu, 0x000001ffu, 0x000003ffu, 0x000007ffu,
					     0x00000fffu, 0x00001fffu, 0x00003fffu, 0x00007fffu, 0x0000ffffu, 0x0001ffffu,
					     0x0003ffffu, 0x0007ffffu, 0x000fffffu, 0x001fffffu, 0x003fffffu, 0x007fffffu,
					     0x00ffffffu, 0x01ffffffu, 0x03ffffffu, 0x07ffffffu, 0x0fffffffu, 0x1fffffffu,
					     0x3fffffffu, 0x7fffffffu, 0xffffffffu};
#endif
<<<<<<< HEAD
*/
