#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include "KernelManager.h"
#include "HGCalRecHitKernelImpl.cuh"

dim3 nblocks_;
constexpr dim3 nthreads_(256); //some kernels will potentially not allocate shared memory properly with a lower number

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit(KernelModifiableData<HGCUncalibratedRecHitSoA, HGCRecHitSoA> *data):
  data_(data)
{
  nblocks_ = (data_->nhits + nthreads_.x - 1) / nthreads_.x;
  nbytes_host_ = (data_->h_out)->nbytes * data_->stride;
  nbytes_device_ = (data_->d_1)->nbytes * data_->stride;
}

KernelManagerHGCalRecHit::~KernelManagerHGCalRecHit()
{
}

void KernelManagerHGCalRecHit::assign_and_transfer_to_device_()
{
  printf("ASSIGN AND TRANSFER:\n");
  printf("%d, %d\n", nbytes_host_, nbytes_device_);
  printf("%d, %d, %d, %d, %d\n", (data_->h_in)->nbytes, (data_->d_1)->nbytes, (data_->d_2)->nbytes, (data_->d_out)->nbytes, (data_->h_out)->nbytes);
  printf("%p\n", (data_->h_in)->amplitude);
  printf("%p\n", (data_->d_1)->amplitude);
  cudaCheck( hipMemcpyAsync((data_->d_1)->amplitude, (data_->h_in)->amplitude, nbytes_device_, hipMemcpyHostToDevice) );
  cudaCheck( hipDeviceSynchronize() ); //needed because the copy is asynchronous
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::assign_and_transfer_to_device_(const KernelConstantData<HGCeeUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGCeeUncalibratedRecHitConstantData> *d_kcdata)
{
  cudaCheck( hipMemcpyAsync( d_kcdata->data.hgcEE_fCPerMIP_, h_kcdata->data.hgcEE_fCPerMIP_, h_kcdata->data.nbytes, hipMemcpyHostToDevice) );
  cudaCheck( hipDeviceSynchronize() ); //needed because the copy is asynchronous
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::assign_and_transfer_to_device_(const KernelConstantData<HGChefUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGChefUncalibratedRecHitConstantData> *d_kcdata)
{
  cudaCheck( hipMemcpyAsync( d_kcdata->data.hgcHEF_fCPerMIP_, h_kcdata->data.hgcHEF_fCPerMIP_, h_kcdata->data.nbytes, hipMemcpyHostToDevice) );
  cudaCheck( hipDeviceSynchronize() ); //needed because the copy is asynchronous
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::assign_and_transfer_to_device_(const KernelConstantData<HGChebUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGChebUncalibratedRecHitConstantData> *d_kcdata)
{
  cudaCheck( hipMemcpyAsync( d_kcdata->data.hgcHEB_fCPerMIP_, h_kcdata->data.hgcHEB_fCPerMIP_, h_kcdata->data.nbytes, hipMemcpyHostToDevice) );
  cudaCheck( hipDeviceSynchronize() ); //needed because the copy is asynchronous
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::transfer_to_host_and_synchronize_()
{
  cudaCheck( hipMemcpyAsync((data_->h_out)->energy, (data_->d_out)->energy, nbytes_host_, hipMemcpyDeviceToHost) );
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::reuse_device_pointers_()
{
  std::swap(data_->d_1, data_->d_2); 
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

LENGTHSIZE KernelManagerHGCalRecHit::get_shared_memory_size_(const LENGTHSIZE& nd, const LENGTHSIZE& nf, const LENGTHSIZE& nu, const LENGTHSIZE& ni, const LENGTHSIZE& nb) {
  LENGTHSIZE dmem = nd*sizeof(double);
  LENGTHSIZE fmem = nf*sizeof(float);
  LENGTHSIZE umem = nu*sizeof(uint32_t);
  LENGTHSIZE imem = ni*sizeof(int);
  LENGTHSIZE bmem = nb*sizeof(bool);
  return dmem + fmem + umem + imem + bmem;
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGCeeUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGCeeUncalibratedRecHitConstantData> *d_kcdata)
{
  assign_and_transfer_to_device_(h_kcdata, d_kcdata);
  assign_and_transfer_to_device_();

  printf("Running ee kernel with: %d hits.\n", data_->nhits);
  printf("%d blocks being launched with %d threads (%d in total).\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x);
  LENGTHSIZE nbytes_shared = get_shared_memory_size_(h_kcdata->data.ndelem, h_kcdata->data.nfelem, h_kcdata->data.nuelem, h_kcdata->data.nielem, h_kcdata->data.nbelem);
  ee_step1<<<nblocks_, nthreads_>>>( *(data_->d_2), *(data_->d_1), d_kcdata->data, data_->nhits );
  after_kernel_();

  //reuse_device_pointers_();

  ee_to_rechit<<<nblocks_, nthreads_, nbytes_shared>>>( *(data_->d_out), *(data_->d_1), d_kcdata->data, data_->nhits );
  after_kernel_();

  transfer_to_host_and_synchronize_();
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChefUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGChefUncalibratedRecHitConstantData> *d_kcdata)
{
  printf("%d blocks being launched with %d threads (%d in total).\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x);
  assign_and_transfer_to_device_(h_kcdata, d_kcdata);
  assign_and_transfer_to_device_();

  printf("Running ee kernel with: %d hits.\n", data_->nhits);
  hef_step1<<<nblocks_,nthreads_>>>( *(data_->d_2), *(data_->d_1), d_kcdata->data, data_->nhits);
  after_kernel_();

  reuse_device_pointers_();

  hef_to_rechit<<<nblocks_,nthreads_>>>( *(data_->d_out), *(data_->d_1), d_kcdata->data, data_->nhits );
  after_kernel_();

  transfer_to_host_and_synchronize_();
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChebUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGChebUncalibratedRecHitConstantData> *d_kcdata)
{
  printf("%d blocks being launched with %d threads (%d in total).\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x);
  assign_and_transfer_to_device_(h_kcdata, d_kcdata);
  assign_and_transfer_to_device_();

  printf("Running ee kernel with: %d hits.\n", data_->nhits);
  heb_step1<<<nblocks_,nthreads_>>>( *(data_->d_2), *(data_->d_1), d_kcdata->data, data_->nhits);
  after_kernel_();

  //reuse_device_pointers_();

  heb_to_rechit<<<nblocks_,nthreads_>>>( *(data_->d_out), *(data_->d_1), d_kcdata->data, data_->nhits );
  after_kernel_();

  transfer_to_host_and_synchronize_();
}

void KernelManagerHGCalRecHit::after_kernel_() {
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

HGCRecHitSoA* KernelManagerHGCalRecHit::get_output()
{
  return data_->h_out;
}
