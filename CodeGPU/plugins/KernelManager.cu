#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "KernelManager.h"
#include "HGCalRecHitKernelImpl.cuh"

dim3 nblocks_;
constexpr dim3 nthreads_(256);

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit(KernelModifiableData<HGCUncalibratedRecHitSoA, HGCRecHitSoA> data, const DetId::Detector& dtype):
  data_(data), dtype_(dtype)
{
  nblocks_ = (data_.nhits + nthreads_.x - 1) / nthreads_.x; 
}

KernelManagerHGCalRecHit::~KernelManagerHGCalRecHit()
{
}

void KernelManagerHGCalRecHit::assign_and_transfer_to_device()
{
  cudaCheck( hipMemcpyAsync((data_.d_1)->amplitude, (data_.h_in)->amplitude, (data_.d_1)->nbytes, hipMemcpyHostToDevice) );
  cudaCheck( hipDeviceSynchronize() ); //needed because the copy is asynchronous
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::transfer_to_host_and_synchronize()
{
  cudaCheck( hipMemcpyAsync((data_.h_out)->energy, (data_.d_out)->energy, (data_.d_out)->nbytes, hipMemcpyDeviceToHost) );
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}


void KernelManagerHGCalRecHit::reuse_device_pointers()
{
  std::swap(data_.d_1, data_.d_2); 
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGCeeUncalibratedRecHitConstantData>& kcdata)
{
  printf("%d blocks being launched with %d threads (%d in total).\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x);
  assign_and_transfer_to_device();

  printf("Running ee kernel with: %zu hits.\n", data_.nhits);
  ee_step1<<<nblocks_,nthreads_>>>( *(data_.d_2), *(data_.d_1), kcdata.data, data_.nhits);
  after_kernel();

  reuse_device_pointers();

  ee_to_rechit<<<nblocks_,nthreads_>>>( *(data_.d_out), *(data_.d_1), kcdata.data, data_.nhits );
  after_kernel();

  //transfer_to_host_and_synchronize();
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChefUncalibratedRecHitConstantData>& kcdata)
{
  printf("%d blocks being launched with %d threads (%d in total).\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x);
  assign_and_transfer_to_device();

  printf("Running ee kernel with: %zu hits.\n", data_.nhits);
  hef_step1<<<nblocks_,nthreads_>>>( *(data_.d_2), *(data_.d_1), kcdata.data, data_.nhits);
  after_kernel();

  reuse_device_pointers();

  hef_to_rechit<<<nblocks_,nthreads_>>>( *(data_.d_out), *(data_.d_1), kcdata.data, data_.nhits );
  after_kernel();

  //transfer_to_host_and_synchronize();
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChebUncalibratedRecHitConstantData>& kcdata)
{
  printf("%d blocks being launched with %d threads (%d in total).\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x);
  assign_and_transfer_to_device();

  printf("Running ee kernel with: %zu hits.\n", data_.nhits);
  heb_step1<<<nblocks_,nthreads_>>>( *(data_.d_2), *(data_.d_1), kcdata.data, data_.nhits);
  after_kernel();

  reuse_device_pointers();

  heb_to_rechit<<<nblocks_,nthreads_>>>( *(data_.d_out), *(data_.d_1), kcdata.data, data_.nhits );
  after_kernel();

  //transfer_to_host_and_synchronize();
}

void KernelManagerHGCalRecHit::after_kernel() {
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

HGCRecHitSoA* KernelManagerHGCalRecHit::get_output()
{
  return data_.h_out;
}
