#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "KernelManager.h"
#include "HGCalRecHitKernelImpl.cuh"

dim3 nblocks_(1);
dim3 nthreads_(32);

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit(KernelManagerData<HGCUncalibratedRecHitSoA, HGCRecHitSoA> data, const DetId::Detector& dtype):
  data_(data), dtype_(dtype)
{
  sbytes_ = shits_ * sizeof(data_.h_in_[0]);
}

KernelManagerHGCalRecHit::~KernelManagerHGCalRecHit()
{
}

void KernelManagerHGCalRecHit::assign_and_transfer_to_device()
{
  cudaCheck( hipMemcpyAsync(data_.d_1_, data_.h_in_, sbytes_, hipMemcpyHostToDevice) );
  cudaCheck( hipDeviceSynchronize() ); //needed because the copy is asynchronous
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::transfer_to_host_and_synchronize()
{
  cudaCheck( hipMemcpyAsync(data_.h_out_, data_.d_out_, sbytes_, hipMemcpyDeviceToHost) );
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::reuse_device_pointers()
{
  //std::swap(data_.d_1_, data_.d_2_); 
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::run_kernels()
{
  assign_and_transfer_to_device();
 
  /*
  if(dtype_ == DetId::HGCalEE)
    {
      ee_step1_wrapper();
      reuse_device_pointers();
      to_rechit_wrapper();
    }
  else if(dtype_ == DetId::HGCalHSi)
    {
      hef_step1_wrapper();
      reuse_device_pointers();
      to_rechit_wrapper();
    }
  else
    {
      heb_step1_wrapper();
      reuse_device_pointers();  
      to_rechit_wrapper();
    }
  */

  transfer_to_host_and_synchronize();
}

void KernelManagerHGCalRecHit::ee_step1_wrapper()
{
  ee_step1<<<nblocks_,nthreads_>>>(data_.d_2_, data_.d_1_, shits_); 
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::hef_step1_wrapper()
{
  hef_step1<<<nblocks_,nthreads_>>>(data_.d_2_, data_.d_1_, shits_); 
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::heb_step1_wrapper()
{  
  hef_step1<<<nblocks_,nthreads_>>>(data_.d_2_, data_.d_1_, shits_); 
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::to_rechit_wrapper()
{
  to_rechit<<<nblocks_,nthreads_>>>(data_.d_out_, data_.d_1_, shits_); 
  cudaCheck( hipGetLastError() );
}

HGCRecHitSoA* KernelManagerHGCalRecHit::get_output()
{
  return data_.h_out_;
}
