#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "KernelManager.h"
#include "HGCalRecHitKernelImpl.cuh"

dim3 nblocks_;
constexpr dim3 nthreads_(256);

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit(KernelManagerData<HGCUncalibratedRecHitSoA, HGCRecHitSoA> data, const DetId::Detector& dtype):
  data_(data), dtype_(dtype)
{
  nblocks_ = (data_.nhits + nthreads_.x - 1) / nthreads_.x; 
  printf("%d blocks being launched with %d threads (%d in total).\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x);
}

KernelManagerHGCalRecHit::~KernelManagerHGCalRecHit()
{
}

void KernelManagerHGCalRecHit::assign_and_transfer_to_device()
{
  printf("%p\n", (data_.h_in)->amplitude);
  printf("Number of bytes copied to GPU: %zu\n", (data_.d_1)->nbytes);
  printf("Address of memory on the GPU: %p\n", (data_.d_1)->amplitude);  
  cudaCheck( hipMemcpyAsync((data_.d_1)->amplitude, (data_.h_in)->amplitude, (data_.d_1)->nbytes, hipMemcpyHostToDevice) );
  cudaCheck( hipDeviceSynchronize() ); //needed because the copy is asynchronous
  cudaCheck( hipGetLastError() );
  printf("%p\n", (data_.h_in)->amplitude);
  printf("Float: %zu, Uint32_t: %zu\n", sizeof(float), sizeof(uint32_t));
  printf("check\n");
}

void KernelManagerHGCalRecHit::transfer_to_host_and_synchronize()
{
  printf("Number of bytes copied to CPU: %zu\n", (data_.d_out)->nbytes);
  cudaCheck( hipMemcpyAsync((data_.h_out)->energy, (data_.d_out)->energy, (data_.d_out)->nbytes, hipMemcpyDeviceToHost) );
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::reuse_device_pointers()
{
  std::swap(data_.d_1, data_.d_2); 
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::run_kernels()
{
  assign_and_transfer_to_device();

  if(dtype_ == DetId::HGCalEE)
    {
      std::cout << "to_rechit() start" << std::endl;
      //ee_step1_wrapper();
      //reuse_device_pointers();
      //to_rechit_wrapper();
      std::cout << "to_rechit() end" << std::endl;
    }
  /*
  else if(dtype_ == DetId::HGCalHSi)
    {
      hef_step1_wrapper();
      reuse_device_pointers();
      to_rechit_wrapper();
    }
  else
    {
      heb_step1_wrapper();
      reuse_device_pointers();  
      to_rechit_wrapper();
    }
  */

  //transfer_to_host_and_synchronize();
}

void KernelManagerHGCalRecHit::ee_step1_wrapper()
{
  printf("Running ee kernel with: %zu hits.\n", data_.nhits);
  ee_step1<<<nblocks_,nthreads_>>>(data_.d_2, data_.d_1, data_.nhits); 
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::hef_step1_wrapper()
{
  hef_step1<<<nblocks_,nthreads_>>>(data_.d_2, data_.d_1, data_.nhits); 
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::heb_step1_wrapper()
{  
  hef_step1<<<nblocks_,nthreads_>>>(data_.d_2, data_.d_1, data_.nhits); 
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::to_rechit_wrapper()
{
  to_rechit<<<nblocks_,nthreads_>>>(data_.d_out, data_.d_1, data_.nhits); 
  cudaCheck( hipGetLastError() );
}

HGCRecHitSoA* KernelManagerHGCalRecHit::get_output()
{
  return data_.h_out;
}
