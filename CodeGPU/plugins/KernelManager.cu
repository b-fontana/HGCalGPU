#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "KernelManager.h"
#include "HGCalRecHitKernelImpl.cuh"

dim3 nblocks_(1);
dim3 nthreads_(32);

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit(const edm::SortedCollection<HGCUncalibratedRecHit>& oldhits):
  oldhits_collection_(oldhits)
{
  shits_ = oldhits_collection_.size();
  if(shits_ == 0)    
    throw cms::Exception("EmptyCollection") << "The collection is empty.";
  for(unsigned int i=0; i<shits_-1; ++i)
    assert(oldhits[i].id().det() == oldhits[i+1].id().det());
      
  DetId detid = oldhits[0].id();
  dtype_ = detid.det(); 
  if( (dtype_ != DetId::HGCalEE) and (dtype_ != DetId::HGCalHSi) and (dtype_ != DetId::HGCalHSc))
    throw cms::Exception("WrongDetectorType") << "The specified detector is wrong.";
    
  sbytes_ = shits_ * sizeof(oldhits_collection_[0]);

  cudaCheck( hipHostMalloc(&h_oldhits_, sbytes_) );
  cudaCheck( hipHostMalloc(&h_newhits_, sbytes_) );
  cudaCheck( hipMalloc((void**)&d_oldhits_, sbytes_) );
  cudaCheck( hipMalloc((void**)&d_newhits_, sbytes_) );
  cudaCheck( hipMalloc((void**)&d_newhits_final_, sbytes_) );
  
  for(unsigned int j=0; j<shits_; ++j)
    h_oldhits_[j] = oldhits_collection_[j];
}

KernelManagerHGCalRecHit::~KernelManagerHGCalRecHit()
{
  cudaCheck( hipHostFree(h_oldhits_) );
  cudaCheck( hipHostFree(h_newhits_) );
  cudaCheck( hipFree(d_oldhits_) );
  cudaCheck( hipFree(d_newhits_) );
}

void KernelManagerHGCalRecHit::assign_and_transfer_to_device()
{
  cudaCheck( hipMemcpyAsync(d_oldhits_, h_oldhits_, sbytes_, hipMemcpyHostToDevice) );
  cudaCheck( hipDeviceSynchronize() ); //needed because the copy is asynchronous
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::transfer_to_host_and_synchronize()
{
  cudaCheck( hipMemcpyAsync(h_newhits_, d_newhits_final_, sbytes_, hipMemcpyDeviceToHost) );
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::reuse_device_pointers()
{
  std::swap(d_oldhits_, d_newhits_);
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::run_kernels()
{
  assign_and_transfer_to_device();
  if(dtype_ == DetId::HGCalEE)
    {
      ee_step1_wrapper();
      reuse_device_pointers();
      to_rechit_wrapper();
    }
  else if(dtype_ == DetId::HGCalHSi)
    {
      hef_step1_wrapper();
      reuse_device_pointers();
      to_rechit_wrapper();
    }
  else
    {
      heb_step1_wrapper();
      reuse_device_pointers();  
      to_rechit_wrapper();
    }
  transfer_to_host_and_synchronize();
}

void KernelManagerHGCalRecHit::ee_step1_wrapper()
{
  ee_step1<<<nblocks_,nthreads_>>>(d_newhits_, d_oldhits_, shits_); 
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::hef_step1_wrapper()
{
  hef_step1<<<nblocks_,nthreads_>>>(d_newhits_, d_oldhits_, shits_); 
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::heb_step1_wrapper()
{  
  hef_step1<<<nblocks_,nthreads_>>>(d_newhits_, d_oldhits_, shits_); 
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::to_rechit_wrapper()
{
  to_rechit<<<nblocks_,nthreads_>>>(d_newhits_final_, d_oldhits_, shits_); 
  cudaCheck( hipGetLastError() );
}

edm::SortedCollection<HGCRecHit> KernelManagerHGCalRecHit::get_new_collection()
{
  return pointer_to_sorted_collection<HGCRecHit>(h_newhits_, shits_);
}
